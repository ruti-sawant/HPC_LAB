
#include <hip/hip_runtime.h>
#include <stdio.h>

void initWith(float num, float *a, int N) {
    for (int i = 0; i < N; ++i) {
        a[i] = num;
    }
}

/*
 * Device kernel stores into `result` the sum of each
 * same-indexed value of `a` and `b`.
 */

__global__ void addVectorsInto(float *result, float *a, float *b, int N) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < N; i += stride) {
        result[i] = a[i] + b[i];
    }
}

void checkElementsAre(float target, float *vector, int N) {
    for (int i = 0; i < N; i++) {
        if (vector[i] != target) {
            printf("FAIL: vector[%d] - %0.0f does not equal %0.0f\n", i, vector[i], target);
            exit(1);
        }
    }
    printf("Success! All values calculated correctly.\n");
}

int main() {
    const int N = 2 << 24;
    size_t size = N * sizeof(float);

    float *a;
    float *b;
    float *c;

    hipMallocManaged(&a, size);
    hipMallocManaged(&b, size);
    hipMallocManaged(&c, size);

    addVectorsInto<<<8, 32>>>(c, a, b, N);
    hipDeviceSynchronize();

    hipFree(a);
    hipFree(b);
    hipFree(c);
}
